#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "timer.h"
#include <math.h>

__global__ void
matrixMultKernel(float* A, float* B, float* C, int n)
{
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    int row = blockIdx.y*blockDim.y + threadIdx.y;

    if (row < n && col < n)
    {
        float val = 0;
        for (int k = 0; k < n; k++)
            val += A[row*n+k] * B[k*n+col];
        C[row*n+col] = val;
    }
}

__global__ void
matVecMultKernel(float* A, float* B, float* C, int n)
{
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    
    if (row < n && col < n)
    {
        val = A[row*n+col] * B[col];
        C[row] = val;
    }
}

void matVecMult(float* A, float* B, float* C, int n)
{
    float val;
    for (int i=0; i<n; i++)
    {
        val = 0;
        for (int j=0; j<n; j++)
        {
            val += A[i*n+j] * B[j];
        }
        C[i] = val;
    }
}

void matrixMult(float* A, float* B, float* C, int n)
{
    float val;
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            val = 0;
            for (int k = 0; k < n; k++)
                val += A[i*n+k] * B[k*n+j];
            C[i*n+j] = val;
        }
    }
}

double sum(float* C, int n)
{
    double s = 0;
    for (int i = 0; i < n; i++)
        for (int j = 0; j < n; j++)
            s += C[i*n+j];
    return s;
}

double vecSum(float* C, int n)
{
    double s = 0;
    for (int i = 0; i < n; i++)
        s += C[i];
    return s;
}

int main(int argc, char* argv[])
{
    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;
    double t0, tfinal;

    int n = atoi(argv[1]);
    int size = n*n*sizeof(float);
    int vecsize=n*sizeof(float);
    int numTests = atoi(argv[2]);

///
    //make file
    FILE * fPtr;
    char fPath[100];
    sprintf(fPath,"Problem2/matvec_N_%d.txt",n);
///DA

    h_A = (float*)malloc(size);
    h_B = (float*)malloc(vecsize);
    h_C = (float*)malloc(vecsize);
    
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, vecsize);
    hipMalloc((void**)&d_C, vecsize);


   for (int test=0;test<numTests;test++)
   {
        for (int i = 0; i < n*n; i++)
        {
            h_A[i] = (float) rand();
            if (i<n) h_B[i] = (float) rand();
        }
        // Matmat
        dim3 dimBlock(32,32);
        int grid_dim = ceil(n / 32.0);
        dim3 dimGrid(grid_dim, grid_dim);
        
//        t0 = get_time();
//        matVecMult(h_A, h_B, h_C, n);
//        tfinal = get_time() - t0;
//        printf("MatVecMult Time %e, Sum %e\n", tfinal, vecSum(h_C, n));

        t0 = get_time();
        hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
        hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
        matVecMultKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, n);
        hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
        tfinal = get_time() - t0;
        printf("MatVecMultKernel Time %e, Size %e\n", tfinal, vecSum(h_C, n));

    ///
        //save time
        fPtr = fopen(fPath ,"a");
        if (fPtr == NULL) exit(EXIT_FAILURE);
        fprintf(fPtr,"%e\n",tfinal);
        fclose(fPtr);
    ///DA

        
   }
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);
    
    return 0;
}

